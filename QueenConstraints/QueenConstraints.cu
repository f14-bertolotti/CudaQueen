#include "hip/hip_runtime.h"
#pragma once
#include "../VariableCollection/VariableCollection.cu"
#include "../ErrorChecking/ErrorChecking.cu"

/*
	constraints to be checked only if VC is all ground
*/
struct DeviceQueenConstraints{

	//////////////////////////////////////SINGLE THREAD//////////////////////////////////////

	__device__ bool static inline checkRowConstraint(DeviceVariableCollection&);		//
	__device__ bool static inline checkColConstraint(DeviceVariableCollection&);		//specific implementation
	__device__ bool static inline checkRDiagConstraint(DeviceVariableCollection&);	//for queen problem
	__device__ bool static inline checkLDiagConstraint(DeviceVariableCollection&);	//
																		
	//////////////////////////////////////MULTI THREAD//////////////////////////////////////

	__device__ bool static inline parallelConstraints(DeviceVariableCollection&);		//specific for queen problem

	////////////////////////////////////////////////////////////////////////////////////////

	__device__ bool static inline solution(DeviceVariableCollection&,bool);			//check solution	
};

///////////////////////////////////////////////////////////////////////////////////////

__device__ bool inline DeviceQueenConstraints::checkRowConstraint(DeviceVariableCollection& vc){

	int sum = 0;
	for(int j = 0; j < vc.nQueen; ++j){
		sum = 0;
		for(int i = 0; i < vc.nQueen; ++i){
			if(vc.deviceVariable[j].domain[i] == 1)++sum;
		}
		if(sum != 1) return false;
	}

	return true;

}

///////////////////////////////////////////////////////////////////////////////////////

__device__ bool inline DeviceQueenConstraints::checkColConstraint(DeviceVariableCollection& vc){

	int sum = 0;
	for(int j = 0; j < vc.nQueen; ++j){
		sum = 0;
		for(int i = 0; i <vc.nQueen; ++i){
			if(vc.deviceVariable[i].domain[j] > 0)++sum;
		}
		if(sum != 1) return false;
	}

	return true;

}

///////////////////////////////////////////////////////////////////////////////////////

__device__ bool inline DeviceQueenConstraints::checkRDiagConstraint(DeviceVariableCollection& vc){

	int sum,i,j,temp;

	for(j = 0; j < vc.nQueen; ++j){
		i = 0;
		sum = 0;
		temp=j;
		while(j < vc.nQueen && i < vc.nQueen){
			if(vc.deviceVariable[i].domain[j]==1)++sum;
			++j;
			++i;
		}
		j = temp;
		if(sum < 0 || sum > 1) return false;
	}

	for(i = 1; i < vc.nQueen; ++i){
		j = 0;
		sum = 0;
		temp = i;
		while(j < vc.nQueen && i < vc.nQueen){
			if(vc.deviceVariable[i].domain[j]==1)++sum;
			++j;
			++i;
		}
		i = temp;
		if(sum < 0 || sum > 1) return false;
	}
	return true;

}

///////////////////////////////////////////////////////////////////////////////////////

__device__ bool inline DeviceQueenConstraints::checkLDiagConstraint(DeviceVariableCollection& vc){

	int sum,i,j,temp;

	for(j = 0; j < vc.nQueen; ++j){
		i = 0;
		sum = 0;
		temp = j;
		while(j >= 0 && i < vc.nQueen){
			if(vc.deviceVariable[i].domain[j]==1)++sum;
			--j;
			++i;
		}
		j = temp;
		if(sum < 0 || sum > 1) return false;
	}

	for(i = 1; i < vc.nQueen; ++i){
		j = vc.nQueen-1;
		sum = 0;
		temp = i;
		while(j >= 0 && i < vc.nQueen){
			if(vc.deviceVariable[i].domain[j]==1)++sum;
			--j;
			++i;
		}
		i = temp;
		if(sum < 0 || sum > 1) return false;
	}

	return true;

}

///////////////////////////////////////////////////////////////////////////////////////

__device__ bool inline DeviceQueenConstraints::solution(DeviceVariableCollection& vc, bool fullParallel){

	if(fullParallel) return parallelConstraints(vc);
	else return checkRowConstraint(vc) && checkColConstraint(vc) && checkRDiagConstraint(vc) && checkLDiagConstraint(vc);

}

///////////////////////////////////////////////////////////////////////////////////////

__global__ void externParallelDiagConstr(int* Mem, int nQueen, bool* okDiags){

	int sum,i,j,what;

	if(threadIdx.x < nQueen)what = 0;
	else if(threadIdx.x >= nQueen && threadIdx.x<2*nQueen)what = 1;
	else if(threadIdx.x >= 2*nQueen && threadIdx.x<3*nQueen)what = 2;
	else if(threadIdx.x >= 3*nQueen && threadIdx.x<4*nQueen)what = 3;

	switch(what){
		case 0:{
			j = threadIdx.x % nQueen;
			i = 0;
			sum = 0;
			while(j < nQueen && i < nQueen){
				if(Mem[nQueen*i + j]==1)++sum;
				++j;
				++i;
			}
			if(sum > 1){
				*okDiags = false;					
			}
			break;
		}
		case 1:{

			i = threadIdx.x % nQueen;
			j = 0;
			sum = 0;
			while(j < nQueen && i < nQueen){
				if(Mem[nQueen*i + j]==1)++sum;
				++j;
				++i;
			}
	
			if(sum > 1){
				*okDiags = false;
			}
			break;
		}
		case 2:{

			j = threadIdx.x % nQueen;
			i = 0;
			sum = 0;
			while(j >= 0 && i < nQueen){
				if(Mem[nQueen*i + j]==1)++sum;
				--j;
				++i;
			}

			if(sum > 1){
				*okDiags = false;
			}
			break;
		}
		case 3:{
			i = threadIdx.x % nQueen;
			j = nQueen-1;
			sum = 0;
			while(j >= 0 && i < nQueen){
				if(Mem[nQueen*i + j]==1)++sum;
				--j;
				++i;
			}
			if(sum > 1){
				*okDiags = false;
			}
			break;
		}
	}

}

__global__ void externParallelAllDiffs(int* Mem, int nQueen, bool* okAllDiffs){

	int sum = 0;
	for(int i = 0 ; i < nQueen; ++i){
		if(Mem[i*nQueen+threadIdx.x]==1)
			++sum;
	}
	
	if(sum != 1){
		*okAllDiffs = false;
	}

}

///////////////////////////////////////////////////////////////////////

__device__ bool inline DeviceQueenConstraints::parallelConstraints(DeviceVariableCollection& vc){

	hipStream_t s1,s2;
	ErrorChecking::deviceErrorCheck(hipStreamCreateWithFlags(&s1, hipStreamNonBlocking),"DeviceQueenConstraints::parallelConstraints::STREAM CREATION 1");
	ErrorChecking::deviceErrorCheck(hipStreamCreateWithFlags(&s2, hipStreamNonBlocking),"DeviceQueenConstraints::parallelConstraints::STREAM CREATION 2");
	__shared__	bool res1, res2;
	res1 = res2 = true;
	externParallelAllDiffs<<<1,vc.nQueen,0,s1>>>(vc.dMem,vc.nQueen,&res1);
	ErrorChecking::deviceErrorCheck(hipPeekAtLastError(),"DeviceQueenConstraints::parallelConstraints::EXTERN PARALLEL ALL DIFFS");
	ErrorChecking::deviceErrorCheck(hipStreamDestroy(s1),"DeviceQueenConstraints::parallelConstraints::STREAM DESTRUCTION 1");
	externParallelDiagConstr<<<1,vc.nQueen*4,0,s2>>>(vc.dMem,vc.nQueen,&res2);
	ErrorChecking::deviceErrorCheck(hipPeekAtLastError(),"DeviceQueenConstraints::parallelConstraints::EXTERN PARALLEL DIAGS DIFFS");
	ErrorChecking::deviceErrorCheck(hipStreamDestroy(s2),"DeviceQueenConstraints::parallelConstraints::STREAM DESTRUCTION 2");
	ErrorChecking::deviceErrorCheck(hipDeviceSynchronize(),"DeviceQueenConstraints::parallelConstraints::SYNCH");
	return res1 && res2;;

}

///////////////////////////////////////////////////////////////////////














