#include "hip/hip_runtime.h"
#include <stdio.h>
#include "./Variable/Variable.cu"
#include "./VariableCollection/VariableCollection.cu"
#include "./QueenConstraints/QueenConstraints.cu"
#include "./QueenPropagation/QueenPropagation.cu"
#include "./TripleQueue/TripleQueue.cu"
#include "./WorkSet/WorkSet.cu"
#include "./ErrorChecking/ErrorChecking.cu"
#include "./parallelQueue/parallelQueue.cu"

////////////////////////////////////////////////////////////////////////////////////////////

__managed__ int nQueen = 13;
__managed__ int maxBlock = 60000;
__managed__ int levelDiscriminant1 = 5;
__managed__ int levelDiscriminant2 = 10;

__device__ int blockCount = 0;
__device__ int nBlockInPhase2 = 0;
__device__ int printLock = 0;
__device__ int solutions = 0;

__device__ DeviceQueenConstraints deviceQueenConstraints;
__device__ DeviceQueenPropagation deviceQueenPropagation;
__device__ DeviceWorkSet deviceWorkSet;
__device__ DeviceParallelQueue deviceParallelQueue;

////////////////////////////////////////////////////////////////////////////////////////////

__global__ void test(int level, int workIndex){

	/*while(atomicCAS(&printLock,0,1)==1){}
	deviceWorkSet.deviceVariableCollection[workIndex].print();
	printLock = 0;*/


	bool done = false;
	int val = 0;

	if(deviceWorkSet.deviceVariableCollection[workIndex].isFailed()){
		return;
	}else if(deviceWorkSet.deviceVariableCollection[workIndex].isGround()){
		if(deviceQueenConstraints.solution(deviceWorkSet.deviceVariableCollection[workIndex],true)){
			atomicAdd(&solutions,1);
			return;
		}
	}

	do{

		if(level < levelDiscriminant1){

			//espansione
			int nExpansion = 0;
			int oldCount = 0;
			nExpansion = deviceWorkSet.expand(workIndex,level,oldCount);

			if(nExpansion >= 0){
				//sono riuscito ad espandere
				deviceWorkSet.deviceVariableCollection[workIndex].deviceQueue.count = 0;

				for(int i = oldCount; i < oldCount + nExpansion; ++i){

				  	hipStream_t s;
				 	hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
				 	deviceWorkSet.deviceVariableCollection[i].lastValues[level] = nQueen;
				 	deviceWorkSet.deviceVariableCollection[i].deviceQueue.count = 0;
					test<<<1,1,0,s>>>(level+1,i);
					hipStreamDestroy(s);

				}

				if(deviceWorkSet.deviceVariableCollection[workIndex].isFailed()){
					done = true;
				}else if(deviceWorkSet.deviceVariableCollection[workIndex].isGround()){
					if(deviceQueenConstraints.solution(deviceWorkSet.deviceVariableCollection[workIndex],true)){
						atomicAdd(&solutions,1);
					}
					done = true;
				}

				++level;
			}else{
				//non sono riuscito ad espandere risolvo normalmente
				atomicAdd(&solutions,deviceWorkSet.solve(workIndex,level));
				done = true;
			}

		}else{

			val = deviceQueenPropagation.nextAssign(deviceWorkSet.deviceVariableCollection[workIndex],level);
			if(val == -1){
				//non sono riuscito ad assegnare
				if(level <= levelDiscriminant1 || level <= 0){
					//ho finito
					done = true;	
				}else{	
					//non ho finito e torno indietro
					deviceQueenPropagation.parallelUndoForwardPropagation(deviceWorkSet.deviceVariableCollection[workIndex]);
					--level;
				}

			}else{
				//sono riuscito ad assegnare e propago
				deviceQueenPropagation.parallelForwardPropagation(deviceWorkSet.deviceVariableCollection[workIndex],level,val);
				
				if(deviceWorkSet.deviceVariableCollection[workIndex].isFailed()){
					//sono in failed e torno indietro
					deviceQueenPropagation.parallelUndoForwardPropagation(deviceWorkSet.deviceVariableCollection[workIndex]);
					--level;
				}else if(deviceWorkSet.deviceVariableCollection[workIndex].isGround()){
					//sono ground e torno indietro
					if(deviceQueenConstraints.solution(deviceWorkSet.deviceVariableCollection[workIndex],true)){
						//se sono anche soluzione incremento solution
						atomicAdd(&solutions,1);
					}
					deviceQueenPropagation.parallelUndoForwardPropagation(deviceWorkSet.deviceVariableCollection[workIndex]);
					--level;
				}

				++level;
			}

		}
/*
		++loop;

	if(workIndex == 0){
		if(loop == 100000){
			done = true;
			printf("done by loop\n");
		}
	}
	if(workIndex > 0){
		if(loop == 100000){
			done = true;
			printf("done by loop\n");
		}
	}
*/
	}while(!done);

	hipDeviceSynchronize();


}

////////////////////////////////////////////////////////////////////////////////////////////

__global__ void initWorkSet( DeviceVariableCollection*,
							 DeviceVariable*,
							 int*,int*,Triple*,int,int);

__global__ void initParallelQueue(DeviceVariableCollection*,
								  DeviceVariable*,
								  int*,int*,int*,int*,Triple*,int,int);

__global__ void results();

////////////////////////////////////////////////////////////////////////////////////////////


int main(){

    hipDeviceSetLimit(hipLimitPrintfFifoSize, sizeof(char)*999999999);
	hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 20);
	hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount,65000);

	HostWorkSet hostWorkSet(nQueen,maxBlock);

	HostParallelQueue hostParallelQueue(nQueen,maxBlock);

	initWorkSet<<<1,1>>>( hostWorkSet.deviceVariableCollection,
						  hostWorkSet.deviceVariable,
						  hostWorkSet.variablesMem,
						  hostWorkSet.lastValuesMem,
						  hostWorkSet.tripleQueueMem,
						  hostWorkSet.nQueen,
						  hostWorkSet.nVariableCollection);

	initParallelQueue<<<1,1>>>(hostParallelQueue.deviceVariableCollection,
				  			   hostParallelQueue.deviceVariable,
				 			   hostParallelQueue.variablesMem,
				 			   hostParallelQueue.lastValuesMem,
				 			   hostParallelQueue.lockReading,
				 			   hostParallelQueue.levelLeaved,
				 			   hostParallelQueue.tripleQueueMem,
				  			   hostParallelQueue.nQueen,
				 			   hostParallelQueue.size);



	hipDeviceSynchronize();
	printf("start\n");

    hipEvent_t     start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
	float   elapsedTime;
	hipEventRecord( start, 0 );

	hipStream_t s;
	ErrorChecking::hostErrorCheck(hipStreamCreateWithFlags(&s, hipStreamNonBlocking),"test::STREAM CREATION");
	test<<<1,1,0,s>>>(0,0);					
	ErrorChecking::hostErrorCheck(hipStreamDestroy(s),"test::STREAM DESTRUCTION");
	ErrorChecking::hostErrorCheck(hipDeviceSynchronize(),"test::SYNCH");
	ErrorChecking::hostErrorCheck(hipPeekAtLastError(),"test::TEST ERROR");

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	results<<<1,1>>>();

	printf("\033[36mTIME: %f\033[0m\n", elapsedTime);

	return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////

__global__ void results(){

	//deviceWorkSet.print();

	printf("\033[32msolutions  = %d\033[0m\n",solutions);
	printf("still in queue = %d\n", deviceParallelQueue.stillInQueue());
	printf("maxUsed = %d\n", deviceParallelQueue.maxUsed);
	printf("block used = %d\n", deviceWorkSet.count);
	printf("block ended = %d\n", nBlockInPhase2);
	printf("block used real = %d\n", temp);
}

////////////////////////////////////////////////////////////////////////////////////////////

__global__ void initWorkSet( DeviceVariableCollection* deviceVariableCollection,
							 DeviceVariable* deviceVariable,
							 int* variablesMem, int* lastValuesMem,
							 Triple* tripleQueueMem, int nQueen, int nVariableCollection){

	deviceWorkSet.init(deviceVariableCollection,
					   deviceVariable,
					   variablesMem,
					   lastValuesMem,
					   tripleQueueMem,
					   nQueen,
					   nVariableCollection);
}

////////////////////////////////////////////////////////////////////////////////////////////

__global__ void initParallelQueue(DeviceVariableCollection* deviceVariableCollection, 
					 			  DeviceVariable* deviceVariable,
					 			  int* variablesMem,
					 			  int* lastValuesMem,
					 			  int* lockReading,
					 			  int* levelLeaved,
					 			  Triple* tripleQueueMem,
					 			  int nQueen, 
					 			  int size){

	deviceParallelQueue.init(deviceVariableCollection,
						     deviceVariable,
						     variablesMem,
						     lastValuesMem,
						     lockReading,
						     levelLeaved,
						     tripleQueueMem,
						     nQueen,
						     size);

}

////////////////////////////////////////////////////////////////////////////////////////////


















