#pragma once
#include "../Variable/Variable2.cu"
#include "../MemoryManagement/MemoryManagement.cu"
#include "../TripleQueue/TripleQueue2.cu"

///////////////////////////////////////////////////////////////////////
////////////////////////HOST SIDE//////////////////////////////////////
///////////////////////////////////////////////////////////////////////

struct HostVariableCollection{
	int* dMem;							//ptr to deviceMemory
	DeviceVariable* dMemVariables;		//vector for variables struct
	int* dMemlastValues;				//last values array
	int nQueen;							//number of variables and also domain size
	bool dbg;							//verbose mode
	HostQueue hostQueue;				//queue

	__host__ HostVariableCollection(int);		//allocate memory with hostMemoryManagemnt
	__host__ ~HostVariableCollection();			//deallocate dMemVariables
};

///////////////////////////////////////////////////////////////////////

__host__ HostVariableCollection::HostVariableCollection(int nq):
	nQueen(nq),dbg(true),hostQueue(nq){

	if(dbg)printf("\033[34mWarn\033[0m::HostVariableCollection::constructor::ALLOCATION\n");
	hipMalloc((void**)&dMemVariables,sizeof(DeviceVariable)*nQueen);
	hipMalloc((void**)&dMemlastValues,sizeof(int)*nQueen);
	hipMalloc((void**)&dMem,sizeof(int)*nQueen*nQueen);
}

///////////////////////////////////////////////////////////////////////

__host__ HostVariableCollection::~HostVariableCollection(){
	if(dbg)printf("\033[34mWarn\033[0m::HostVariableCollection::destructor::DELLOCATION\n");
	hipFree(dMemVariables);
	hipFree(dMemlastValues);
	hipFree(dMem);
}

///////////////////////////////////////////////////////////////////////
////////////////////////DEVICE SIDE////////////////////////////////////
///////////////////////////////////////////////////////////////////////

struct DeviceVariableCollection{

	bool dbg;					//verbose
	bool fullParallel;			//chose parallel code
	int nQueen;					//number of variables and domain size
	int* lastValues;			//last values array
	DeviceVariable* variables;	//array for variables
	DeviceQueue deviceQueue;	//triple queue
	DeviceMemoryManagement deviceMemoryManagement;	
						//structure for fast modification
						//of the memory

	__device__ DeviceVariableCollection();											//do nothing
	__device__ DeviceVariableCollection(DeviceVariable*,Triple*, int*,int*,int);	//initialize
	__device__ void init(DeviceVariable*,Triple*,int*,int*,int);					//initialize
	__device__ ~DeviceVariableCollection();											//do nothing

	__device__ bool isGround();			//check if every variable is not failed
	__device__ bool isFailed();			//check if every variable is ground

	__device__ void print();			//print collection

};

///////////////////////////////////////////////////////////////////////

__device__ DeviceVariableCollection::DeviceVariableCollection(){}

///////////////////////////////////////////////////////////////////////

__device__ DeviceVariableCollection::DeviceVariableCollection(DeviceVariable* dv,Triple* q, int* vm, int* lv, int nq):
	dbg(true),fullParallel(true),nQueen(nq),variables(dv),
	deviceMemoryManagement(vm,1,nQueen,nQueen),deviceQueue(q,nq),lastValues(lv){
	
	if(fullParallel)deviceMemoryManagement.setMatrixFromToMultiLess(0,0,1);
	else deviceMemoryManagement.setMatrix(0,1);
	for (int i = 0; i < nQueen; ++i){
		variables[i].init2(&vm[nQueen*i],nQueen);
		lastValues[i]=0;
	}
}

///////////////////////////////////////////////////////////////////////

__device__ void DeviceVariableCollection::init(DeviceVariable* dv,Triple* q, int* vm, int* lv, int nq){
	dbg = true;
	fullParallel = true;
	nQueen = nq;
	variables = dv;
	lastValues = lv;
	deviceQueue.init(q,nq);
	deviceMemoryManagement.init(vm,1,nQueen,nQueen);
	if(fullParallel)deviceMemoryManagement.setMatrixFromToMultiLess(0,0,1);
	else deviceMemoryManagement.setMatrix(0,1);
	for (int i = 0; i < nQueen; ++i){
		variables[i].init2(&vm[nQueen*i],nQueen);
		lastValues[i]=0;
	}
}

///////////////////////////////////////////////////////////////////////

__device__ DeviceVariableCollection::~DeviceVariableCollection(){}

///////////////////////////////////////////////////////////////////////

__device__ void DeviceVariableCollection::print(){
	for (int i = 0; i < nQueen; ++i){
		printf("[%d] ::: ",lastValues[i]);
		variables[i].print();
	}
	deviceQueue.print();
	printf("\n");
}

///////////////////////////////////////////////////////////////////////

__device__ bool DeviceVariableCollection::isGround(){
	for(int i = 0; i < nQueen; ++i)
		if(variables[i].ground==-1)return false;

	return true;
}

///////////////////////////////////////////////////////////////////////

__device__ bool DeviceVariableCollection::isFailed(){
	for(int i = 0; i < nQueen; ++i)
		if(variables[i].failed == 1)return true;

	return false;
}

///////////////////////////////////////////////////////////////////////