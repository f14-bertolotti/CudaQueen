#include "hip/hip_runtime.h"
#pragma once
#include "../Variable/Variable.cu"
#include "../TripleQueue/TripleQueue.cu"
#include "../VariableCollection/VariableCollection.cu"
#include "../QueenPropagation/QueenPropagation.cu"
#include "../QueenConstraints/QueenConstraints.cu"
#include "../ErrorChecking/ErrorChecking.cu"
#include <cstdio>

///////////////////////////////////////////////////////////////////////
////////////////////////HOST SIDE//////////////////////////////////////
///////////////////////////////////////////////////////////////////////

struct HostParallelQueue{

	DeviceVariableCollection* deviceVariableCollection;

	DeviceVariable* deviceVariable;
	int* variablesMem;
	int* lastValuesMem;
	int* lockReading;
	int* levelLeaved;
	Triple* tripleQueueMem;

	int size;
	int nQueen;

	__host__ HostParallelQueue(int,int);
	__host__ ~HostParallelQueue();
};

//////////////////////////////////////////////////////////////////////////////////////////////

__host__ HostParallelQueue::HostParallelQueue(int nq, int sz):nQueen(nq),size(sz){
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&lockReading,sizeof(int)*size),"Error::HostParallelQueue::ALLOCATE 1");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&deviceVariableCollection,sizeof(DeviceVariableCollection)*size),"Error::HostParallelQueue::ALLOCATE 2");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&levelLeaved,sizeof(int)*size),"Error::HostParallelQueue::ALLOCATE 3");

	ErrorChecking::hostErrorCheck(hipMalloc((void**)&deviceVariable,sizeof(DeviceVariable)*size*nQueen),"HostParallelQueue::DEVICE VARIABLE ALLOCATION");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&variablesMem,sizeof(int)*nQueen*nQueen*size),"HostParallelQueue::VARIABLE MEM ALLOCATION");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&lastValuesMem,sizeof(int)*nQueen*size),"HostParallelQueue::LAST VALUES MEM ALLOCATION");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&tripleQueueMem,sizeof(Triple)*nQueen*nQueen*3*size),"HostParallelQueue::TRIPLE QUEUE MEM ALLOCATION");

	ErrorChecking::hostErrorCheck(hipPeekAtLastError(),"HostParallelQueue::EXTERN SET CALL");
	ErrorChecking::hostErrorCheck(hipDeviceSynchronize(),"HostParallelQueue::SYNCH");

}

//////////////////////////////////////////////////////////////////////////////////////////////

__host__ HostParallelQueue::~HostParallelQueue(){
	ErrorChecking::hostErrorCheck(hipFree(levelLeaved),"Error::hostParallelQueue::DEALLOCATE 1");
	ErrorChecking::hostErrorCheck(hipFree(lockReading),"Error::hostParallelQueue::DEALLOCATE 1");
	ErrorChecking::hostErrorCheck(hipFree(deviceVariableCollection),"Error::hostParallelQueue::DEALLOCATE 2");
	ErrorChecking::hostErrorCheck(hipFree(variablesMem),"Error::hostParallelQueue::VARIABLES MEM DEALLOCATION");
	ErrorChecking::hostErrorCheck(hipFree(lastValuesMem),"Error::hostParallelQueue::LAST VALUES MEM DEALLOCATION");
	ErrorChecking::hostErrorCheck(hipFree(tripleQueueMem),"Error::hostParallelQueue::TRIPLE QUEUE ME DEALLOCATION");
	ErrorChecking::hostErrorCheck(hipFree(deviceVariable),"Error::hostParallelQueue::DEVICE VARIABLE DEALLOCATION");
}

///////////////////////////////////////////////////////////////////////
////////////////////////DEVICE SIDE////////////////////////////////////
///////////////////////////////////////////////////////////////////////

struct DeviceParallelQueue{
	int size;									//max number of element(fixed)
	int nQueen;									//size of csp

	DeviceVariableCollection* deviceVariableCollection;
	DeviceVariable* deviceVariable;
	int* lockReading;
	int* variablesMem;
	int* lastValuesMem;
	int* levelLeaved;
	Triple* tripleQueueMem;

	__device__ DeviceParallelQueue();					//do nothing
	__device__ DeviceParallelQueue(DeviceVariableCollection*,DeviceVariable*,int*,int*,int*,int*,Triple*,int,int);	//initialize
	__device__ void init(DeviceVariableCollection*,DeviceVariable*,int*,int*,int*,int*,Triple*,int,int);			//initialize

	__device__ int add(DeviceVariableCollection&,int,int);	//add an element, -1 if fail
	__device__ int read(DeviceVariableCollection&,int);		//returns last and delete last element, -1 if fail

	__device__ void print();					//print
	__device__ void printLocks();

	__device__ ~DeviceParallelQueue();				//do nothing
};

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ DeviceParallelQueue::DeviceParallelQueue(){}

//////////////////////////////////////////////////////////////////////////////////////////////

__global__ void ParallelQueueExternInit(DeviceVariableCollection* deviceVariableCollection,
									    DeviceVariable* deviceVariable, int* variablesMem,
									    int* lastValuesMem, int* lockReading, Triple* tripleQueueMem,
									    int nQueen, int nVariableCollection){

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < nQueen*nVariableCollection){

		deviceVariable[index].init2(&variablesMem[index*nQueen],nQueen);

		if(index < nVariableCollection){

			deviceVariableCollection[index].init2(&deviceVariable[index*nQueen],
												 &tripleQueueMem[index*nQueen*nQueen*3],
												 &variablesMem[index*nQueen*nQueen],
												 &lastValuesMem[index*nQueen],nQueen);

		}

	}

	if(index < nVariableCollection)
		lockReading[index] = 0;
}

__device__ DeviceParallelQueue::DeviceParallelQueue(DeviceVariableCollection* dvc, 
													DeviceVariable* dv,
													int* vm, int* lvm, int* lr, int* ll,
													Triple* tqm,
													int nq, int sz):
													deviceVariableCollection(dvc),deviceVariable(dv),
													variablesMem(vm),levelLeaved(ll),lastValuesMem(lvm),tripleQueueMem(tqm),
													lockReading(lr),nQueen(nq),size(sz){

	ParallelQueueExternInit<<<int(size*nQueen)/1000+1,1000>>>(deviceVariableCollection,
															  deviceVariable,
															  variablesMem,
											    			  lastValuesMem,
											    			  lockReading,
															  tripleQueueMem,
															  nQueen,size);
	ErrorChecking::deviceErrorCheck(hipPeekAtLastError(),"DeviceParallelQueue::DeviceParallelQueue::EXTERN INIT");


}

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ void DeviceParallelQueue::init(DeviceVariableCollection* dvc, DeviceVariable* dv,
								 	  int* vm, int* lvm, int* lr, int* ll, Triple* tqm, int nq, int sz){

	variablesMem = vm;
	lastValuesMem = lvm;
	tripleQueueMem = tqm;

	deviceVariable = dv;
	deviceVariableCollection = dvc;

	lockReading = lr;
	levelLeaved = ll;

	nQueen = nq;
	size = sz;

	ParallelQueueExternInit<<<int(size*nQueen)/1000+1,1000>>>(deviceVariableCollection,
											 				  deviceVariable,
											 				  variablesMem,
							    			 				  lastValuesMem,
							    			 				  lockReading,
											 				  tripleQueueMem,
											 				  nQueen,size);
	ErrorChecking::deviceErrorCheck(hipPeekAtLastError(),"DeviceParallelQueue::init::EXTERN INIT");
}

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ int DeviceParallelQueue::add(DeviceVariableCollection& element, int level, int index){

	int pos = -1;
	for (int i = 0; i < size; ++i){
		if(atomicCAS(&lockReading[i],0,1)==0){
			pos = i;
			break;
		}
	}
	if(pos == -1)return -1;

	levelLeaved[pos] = level;
	deviceVariableCollection[pos] = element;

	lockReading[pos] = 2;

	return 0;
}

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ int DeviceParallelQueue::read(DeviceVariableCollection& element, int index){

	int pos = -1;
	for (int i = 0; i < size; ++i){
		if(atomicCAS(&lockReading[i],2,3)==2){
			pos = i;
			break;
		}
	}

	if(pos == -1)return -1;
	element = deviceVariableCollection[pos];

	lockReading[pos] = 0;

	return levelLeaved[pos];
}

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ void DeviceParallelQueue::print(){

	for(int i = 0; i < size; ++i) {
		printf("------[%d,%s,%d]------\n", i, lockReading[i] ? "locked" : "free",levelLeaved[i]);
		deviceVariableCollection[i].print();
	}

	printf("size:%d\n",size);
}

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ void DeviceParallelQueue::printLocks(){
	for(int i = 0; i < size; ++i){
		if(i % 100 == 0)printf("\n");
		printf("%d", lockReading[i]);
	}printf("\n");
}

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ DeviceParallelQueue::~DeviceParallelQueue(){}

//////////////////////////////////////////////////////////////////////////////////////////////
