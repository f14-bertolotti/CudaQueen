#include "hip/hip_runtime.h"
#pragma once
#include "../VariableCollection/VariableCollection.cu"
#include "../ErrorChecking/ErrorChecking.cu"

struct DeviceQueenPropagation{

	//////////////////////////////////////SINGLE THREAD//////////////////////////////////////

	__device__ int static inline nextAssign(DeviceVariableCollection&,int);		//assign next value not already tried
																	//returns assigned value

	__device__ int static inline allDifferent(DeviceVariableCollection&,int,int,int);		//propagate for all different constraint code 3
	__device__ int static inline diagDifferent(DeviceVariableCollection&,int,int,int);	//propagate for diag constraint code 4

	__device__ int static inline forwardPropagation(DeviceVariableCollection&,int,int);	//csp forward propagation code 5
	__device__ int static inline undoForwardPropagation(DeviceVariableCollection&);		//csp undo forward propagation

	//////////////////////////////////////MULTI THREAD//////////////////////////////////////

	__device__ int static inline parallelPropagation(DeviceVariableCollection&,int,int,int); 		//propagation multithread code 2
	__device__ int static inline parallelForwardPropagation(DeviceVariableCollection&,int,int);	//forward, uses parallelPropagation code 5
	__device__ int static inline parallelUndoForwardPropagation(DeviceVariableCollection&);		//csp undo forward propagation

};

////////////////////////////////////////////////////////////////////////////

__device__ int inline DeviceQueenPropagation::nextAssign(DeviceVariableCollection& vc, int var){

	if(var < 0 || var >= vc.nQueen){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::nextAssign::VAR OUT OF BOUND");
		return -1;
	}

	if(vc.lastValues[var] >= vc.nQueen){
		ErrorChecking::deviceMessage("Warn::DeviceQueenPropagation::nextAssign::VALUE OUT OF BOUND");
		return -1;
	}

	if(vc.deviceVariable[var].failed == 1){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::nextAssign::VAR ALREADY FAILED");
		return -1;
	}

	int next;
	for(next = vc.lastValues[var];next<vc.nQueen;++next)
		if(vc.deviceVariable[var].domain[next]==1){
			vc.lastValues[var]=next+1;
			vc.deviceVariable[var].assign(next);
			return next;
		}

	ErrorChecking::deviceMessage("Warn::DeviceQueenPropagation::nextAssign::NEXTVALUE NOT FOUND");

	return -1;

}

////////////////////////////////////////////////////////////////////////////

__device__ int inline DeviceQueenPropagation::allDifferent(DeviceVariableCollection& vc, int var, int val, int delta){

	if(var < 0 || var > vc.nQueen || val < 0 || val > vc.nQueen){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::allDifferent::OUT OF BOUND");
		return -1;
	}

	if(vc.deviceVariable[var].ground != val){
		ErrorChecking::deviceError("Error::QueenPropagation::allDifferent::VARIABLE NOT GROUND");
		return -1;
	}
	
	for(int i = 0; i < vc.nQueen; ++i)
		if(i != var){
			vc.deviceVariable[i].addTo(val,delta);

		}
	
	if(delta < 0)vc.deviceQueue.add(var,val,3);

	return 0;	

}

////////////////////////////////////////////////////////////////////////////

__device__ int inline DeviceQueenPropagation::diagDifferent(DeviceVariableCollection& vc, int var, int val, int delta){

	if(var < 0 || var > vc.nQueen || val < 0 || val > vc.nQueen){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::diagDifferent::OUT OF BOUND");
		return -1;
	}

	if(vc.deviceVariable[var].ground != val){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::diagDifferent::VARIABLE NOT GROUND");
		return -1;
	}

	int i=var+1,j=val+1;
	while(i<vc.nQueen && j<vc.nQueen){
		vc.deviceVariable[i].addTo(j,delta);
		++i;++j;
	}

	i=var-1,j=val-1;
	while(i>=0 && j>=0){
		vc.deviceVariable[i].addTo(j,delta);
		--i;--j;
	}

	i=var-1,j=val+1;
	while(i>=0 && j<vc.nQueen){
		vc.deviceVariable[i].addTo(j,delta);
		--i;++j;
	}

	i=var+1,j=val-1;
	while(i<vc.nQueen && j>=0){
		vc.deviceVariable[i].addTo(j,delta);
		++i;--j;
	}

	if(delta < 0)vc.deviceQueue.add(var,val,4);
	return 0;

}

////////////////////////////////////////////////////////////////////////////

__device__ int inline DeviceQueenPropagation::forwardPropagation(DeviceVariableCollection& vc, int var, int val){

	if(var < 0 || var > vc.nQueen){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::forwardPropagation:: VAR OUT OF BOUND");
		return -1;
	}

	if(val < 0 || val > vc.nQueen){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::forwardPropagation:: VAL OUT OF BOUND");
		return -1;
	}

	if(vc.deviceVariable[var].ground != val){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::forwardPropagation::VARIABLE NOT GROUND");
		return -1;
	}

	allDifferent(vc,var,val,-1);
	diagDifferent(vc,var,val,-1);

	bool ch = false;
	do{
		ch=false;
		for(int i = 0; i < vc.nQueen; ++i){
			if(vc.deviceVariable[i].changed==1){
				if(vc.deviceVariable[i].ground>=0){
					allDifferent(vc,i,vc.deviceVariable[i].ground,-1);
					diagDifferent(vc,i,vc.deviceVariable[i].ground,-1);
					ch = true;
				}
				vc.deviceVariable[i].changed=-1;
			}
		}
	}while(ch);

	vc.deviceQueue.add(var,val,5);

	return 0;

}

////////////////////////////////////////////////////////////////////////////

__device__ int inline DeviceQueenPropagation::undoForwardPropagation(DeviceVariableCollection& vc){

	if(vc.deviceQueue.front()->cs!=5){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::undoForwardPropagation::ERROR IN QUEUE");
		return -1;		
	}

	if(vc.deviceQueue.empty()){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::undoForwardPropagation::EMPTY QUEUE");
		return -1;		
	}

	int t1=vc.deviceQueue.front()->var;
	int t2=vc.deviceQueue.front()->val;

	for(int i = t1+1; i < vc.nQueen; ++i)vc.lastValues[i]=0;

	vc.deviceQueue.pop();
	while(vc.deviceQueue.front()->cs!=5){
		switch(vc.deviceQueue.front()->cs){
			case 3:{
				allDifferent(vc,vc.deviceQueue.front()->var,vc.deviceQueue.front()->val,+1);	
			}break;
			case 4:{
				diagDifferent(vc,vc.deviceQueue.front()->var,vc.deviceQueue.front()->val,+1);	
			}break;
		}
		vc.deviceQueue.pop();

		if(vc.deviceQueue.empty())break;
	}

	vc.deviceVariable[t1].undoAssign(t2);
	return 0;

}

////////////////////////////////////////////////////////////////////////////

__global__ void externPropagation(DeviceVariableCollection& vc, int var, int val, int nQueen,int delta){

	int col = int((threadIdx.x + blockIdx.x * blockDim.x % (nQueen * nQueen))%nQueen);
	int row = int(((threadIdx.x + blockIdx.x * blockDim.x % (nQueen * nQueen))/nQueen) % nQueen);

	if(row != var && val == col)
		vc.deviceVariable[row].addTo(col,delta);
	
	
	if(row != var && col == row && col+val-var < nQueen && col+val-var >= 0)
		vc.deviceVariable[row].addTo(col+val-var,delta);
	

	if(row != var && nQueen-col == row && col-(nQueen-val)+var < nQueen && col-(nQueen-val)+var >= 0)
		vc.deviceVariable[row].addTo(col-(nQueen-val)+var,delta);

}

__device__ int inline DeviceQueenPropagation::parallelPropagation(DeviceVariableCollection& vc,int var,int val,int delta){

	if(var < 0 || var > vc.nQueen || val < 0 || val > vc.nQueen){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::parallelPropagation::OUT OF BOUND");
		return -1;
	}

	if(vc.deviceVariable[var].ground != val){
		ErrorChecking::deviceError("Error::QueenPropagation::parallelPropagation::VARIABLE NOT GROUND");
		return -1;
	}

	hipStream_t s;
	ErrorChecking::deviceErrorCheck(hipStreamCreateWithFlags(&s, hipStreamNonBlocking),"DeviceQueenPropagation::parallelPropagation::STREAM CREATION");
	externPropagation<<<1,vc.nQueen*vc.nQueen,0,s>>>(vc,var,val,vc.nQueen,delta);
	ErrorChecking::deviceErrorCheck(hipPeekAtLastError(),"DeviceQueenPropagation::parallelPropagation::EXTERN PROPAGATION CALL");
	ErrorChecking::deviceErrorCheck(hipStreamDestroy(s),"DeviceQueenPropagation::parallelPropagation::STREAM DESTRUCTION");
	if(delta < 0)vc.deviceQueue.add(var,val,6);
	ErrorChecking::deviceErrorCheck(hipDeviceSynchronize(),"DeviceQueenPropagation::parallelPropagation::SYNCH");

	return 0;

}

////////////////////////////////////////////////////////////////////////////

__device__ int inline DeviceQueenPropagation::parallelForwardPropagation(DeviceVariableCollection& vc, int var, int val){

	if(var < 0 || var > vc.nQueen){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::parallelForwardPropagation::VAR OUT OF BOUND");
		return -1;
	}

	if(val < 0 || val > vc.nQueen){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::parallelForwardPropagation::VAL OUT OF BOUND");
		return -1;
	}

	if(vc.deviceVariable[var].ground != val){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::parallelForwardPropagation::VARIABLE NOT GROUND");
		return -1;
	}

	parallelPropagation(vc,var,val,-1);

	bool ch = false;
	do{
		ch=false;
		for(int i = 0; i < vc.nQueen; ++i){
			if(vc.deviceVariable[i].changed==1){
				if(vc.deviceVariable[i].ground>=0){
					parallelPropagation(vc,i,vc.deviceVariable[i].ground,-1);
					ch = true;
				}
				vc.deviceVariable[i].changed=-1;
			}
		}
	}while(ch);

	vc.deviceQueue.add(var,val,5);

	return 0;
}

////////////////////////////////////////////////////////////////////////////

__device__ int inline DeviceQueenPropagation::parallelUndoForwardPropagation(DeviceVariableCollection& vc){

	if(vc.deviceQueue.front()->cs!=5){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::parallelUndoForwardPropagation::ERROR IN QUEUE");
		return -1;		
	}

	if(vc.deviceQueue.empty()){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::parallelUndoForwardPropagation::EMPTY QUEUE");
		return -1;		
	}

	int t1=vc.deviceQueue.front()->var;
	int t2=vc.deviceQueue.front()->val;

	for(int i = t1+1; i < vc.nQueen; ++i)vc.lastValues[i]=0;

	vc.deviceQueue.pop();
	while(vc.deviceQueue.front()->cs!=5){
		hipStream_t s;
		ErrorChecking::deviceErrorCheck(hipStreamCreateWithFlags(&s, hipStreamNonBlocking),"DeviceQueenPropagation::parallelUndoForwardPropagation::STREAM CREATION");
		externPropagation<<<1,vc.nQueen*vc.nQueen,0,s>>>(vc,vc.deviceQueue.front()->var,vc.deviceQueue.front()->val,vc.nQueen,+1);
		ErrorChecking::deviceErrorCheck(hipPeekAtLastError(),"DeviceQueenPropagation::parallelUndoForwardPropagation::EXTERN PROPAGATION CALL");
		ErrorChecking::deviceErrorCheck(hipStreamDestroy(s),"DeviceQueenPropagation::parallelUndoForwardPropagation::STREAM DESTRUCTION");
		ErrorChecking::deviceErrorCheck(hipDeviceSynchronize(),"DeviceQueenPropagation::parallelUndoForwardPropagation::STREAM SYNCH");
		vc.deviceQueue.pop();
		if(vc.deviceQueue.empty())break;
	}

	vc.deviceVariable[t1].undoAssign(t2);

	return 0;
}

////////////////////////////////////////////////////////////////////////////

