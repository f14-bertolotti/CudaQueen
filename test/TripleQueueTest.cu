#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../TripleQueue/TripleQueue.cu"

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ DeviceQueue deviceQueue;

//////////////////////////////////////////////////////////////////////////////////////////////

__global__ void init(Triple*,int);
__global__ void test();

//////////////////////////////////////////////////////////////////////////////////////////////

int main(){
	HostQueue hostQueue(8);
	init<<<1,1>>>(hostQueue.dMem, hostQueue.nQueen);
	test<<<1,1>>>();
	hipDeviceSynchronize();
	return 0;
}

//////////////////////////////////////////////////////////////////////////////////////////////

__global__ void init(Triple* dMem, int nQueen){
	deviceQueue.init(dMem, nQueen);
}

//////////////////////////////////////////////////////////////////////////////////////////////

__global__ void test(){

	deviceQueue.add(1,1,1);
	deviceQueue.add(2,1,1);
	deviceQueue.add(3,1,1);
	deviceQueue.add(1,1,5);
	deviceQueue.add(1,1,1);
	deviceQueue.add(2,1,1);
	deviceQueue.add(3,1,1);				
	deviceQueue.add(1,1,5);
	Triple t = *deviceQueue.front();
	deviceQueue.print();
	printf("----%d,%d,%d----\n", t.var,t.val,t.cs);
	deviceQueue.pop();
	deviceQueue.pop();
	deviceQueue.pop();
	deviceQueue.pop();
	deviceQueue.print();

}

//////////////////////////////////////////////////////////////////////////////////////////////
/*
Warn::HostQueue::constructor::ALLOCATION
(1,1,1)
(2,1,1)
(3,1,1)
(1,1,5)
(1,1,1)
(2,1,1)
(3,1,1)
(1,1,5)
----1,1,5----
(1,1,1)
(2,1,1)
(3,1,1)
(1,1,5)
Warn::HostQueue::destructor::DELLOCATION
*/
//////////////////////////////////////////////////////////////////////////////////////////////
